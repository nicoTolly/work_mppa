#include "hip/hip_runtime.h"
#include "loop_kernel.hu"
__global__ void kernel0(float *dst, float *tab)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 <= 1999)
      for (int c3 = t1; c3 <= ppcg_min(31, -32 * b1 + 1999); c3 += 16)
        dst[64000 * b0 + 32 * b1 + 2000 * t0 + c3] = (tab[64000 * b0 + 32 * b1 + 2000 * t0 + c3] + tab[64000 * b0 + 32 * b1 + 2000 * t0 + c3]);
}
