#include <assert.h>
#include <stdio.h>
#include "loop_kernel.hu"
#include <stdio.h>
#include <stdlib.h>
#define N 2000
int main()
{
	float tab[N*N];
	float dst[N*N];
	float sum;
	for(int i = 0; i < N ; i++)
		for(int j = 0; j < N ; j++)
			tab[i*N + j] = 1.2 + i * 3.5;

	{
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

	  float *dev_dst;
	  float *dev_tab;
	  
	  cudaCheckReturn(hipMalloc((void **) &dev_dst, (4000000) * sizeof(float)));
	  cudaCheckReturn(hipMalloc((void **) &dev_tab, (4000000) * sizeof(float)));
	  
	  cudaCheckReturn(hipMemcpy(dev_tab, tab, (4000000) * sizeof(float), hipMemcpyHostToDevice));
	  {
	    dim3 k0_dimBlock(16, 32);
	    dim3 k0_dimGrid(63, 63);
	    kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_dst, dev_tab);
	    cudaCheckKernel();
	  }
	  
	  cudaCheckReturn(hipMemcpy(dst, dev_dst, (4000000) * sizeof(float), hipMemcpyDeviceToHost));
	  cudaCheckReturn(hipFree(dev_dst));
	  cudaCheckReturn(hipFree(dev_tab));
	}
	sum = 0;
	for(int i = 0; i < N * N ; i++)
		sum += dst[i];
	printf("%f\n", sum);
	return 0;
}
